#include "hip/hip_runtime.h"
#include "blast.h"
#include <semiGappedScoringGPU.cu>
#include <sys/time.h>


void getAlignmentsNum(int *alignmentsNum,
					  int *ungappedExtensionNum)
{
	struct alignment *alignment;
	struct ungappedExtension *ungappedExtension;
	int alignmentsNo, ungappedExtensionNo, totalUngappedNum = 0;
	alignmentsNo = 0;
	memBlocks_resetCurrent(alignments_alignments);
	while ((alignment = (struct alignment *)memBlocks_getCurrent(alignments_alignments)) != NULL)
	{
		alignmentsNo++;
		ungappedExtensionNo = 0;
		ungappedExtension = alignment->ungappedExtensions;
		while (ungappedExtension != NULL)
		{
			ungappedExtensionNo++;
			ungappedExtension = ungappedExtension->next;
		}
		totalUngappedNum += ungappedExtensionNo;
	}

	*alignmentsNum = alignmentsNo;
	*ungappedExtensionNum = totalUngappedNum;

	return;
}


//void copyAlignmentsToBuffer(struct alignment *alignmentArray,
void copyAlignmentsToBuffer(struct ungappedExtension *ungappedExtensionArray,
							int *ungappedExtensionStartLoc,
							int *ungappedExtensionNum)
{
	struct alignment *alignment;
	struct ungappedExtension *ungappedExtension;
	struct ungappedExtension *curUngappedExtension;

	int alignmentNo, ungappedExtensionNo;
	int ungappedStartLoc = 0;
	alignmentNo = 0;
	memBlocks_resetCurrent(alignments_alignments);
	while ((alignment = (struct alignment *)memBlocks_getCurrent(alignments_alignments)) != NULL)
	{
		ungappedExtensionStartLoc[alignmentNo] = ungappedStartLoc;
		curUngappedExtension = &ungappedExtensionArray[ungappedStartLoc];
		ungappedExtension = alignment->ungappedExtensions;
		ungappedExtensionNo = 0;
		while (ungappedExtension != NULL)
		{
			memcpy(curUngappedExtension + ungappedExtensionNo, ungappedExtension, sizeof(struct ungappedExtension));
			ungappedExtensionNo++;
			ungappedExtension = ungappedExtension->next;
		}

		ungappedExtensionNum[alignmentNo] = ungappedExtensionNo;
		ungappedStartLoc += ungappedExtensionNo;
		alignmentNo++;
	}
}

void updataUngappedExtensions(struct ungappedExtension *ungappedExtensionArray)
{
	struct alignment *alignment;
	struct ungappedExtension *ungappedExtension;
	struct ungappedExtension *curUngappedExtension;

	int alignmentNo, ungappedExtensionNo = 0;
	memBlocks_resetCurrent(alignments_alignments);
	while ((alignment = (struct alignment *)memBlocks_getCurrent(alignments_alignments)) != NULL)
	{
		ungappedExtension = alignment->ungappedExtensions;
		while (ungappedExtension != NULL)
		{
			memcpy(ungappedExtension, ungappedExtensionArray + ungappedExtensionNo, sizeof(struct ungappedExtension));
			ungappedExtensionNo++;
			ungappedExtension = ungappedExtension->next;
		}
	}

	return;
}

void addGoodAlignment(int4 *bestScoreArray, 
					  int4 *numGoodExtensions, 
					  int4 *numSemiGapping)
{
	struct alignment *alignment;
	int alignmentNo = 0;
	memBlocks_resetCurrent(alignments_alignments);
	while ((alignment = (struct alignment *)memBlocks_getCurrent(alignments_alignments)) != NULL)
	{
		if (bestScoreArray[alignmentNo] >= blast_nominalR1cutoff)
		{
			alignments_addGoodAlignment(bestScoreArray[alignmentNo], alignment);
			blast_numGoodExtensions += numGoodExtensions[alignmentNo];
			blast_numGoodAlignments++;
		}

		blast_numSemiGapped += numSemiGapping[alignmentNo];

		alignmentNo++;
	}
	
	return;
}

//sort the alignment according to the num of ungapped extensions contained.
void sortAlignments(int *ungappedExtensionNum, 
					int *orderArray,
					int nBegin, int nEnd)
{
	int piv, l, r, temp;

	if (nEnd > nBegin + 1)
	{
		piv = ungappedExtensionNum[nBegin];
		l = nBegin + 1;
		r = nEnd;

		while (l < r)
		{
			if (ungappedExtensionNum[l] <= piv)
			{
				l++;
			}
			else
			{
				--r;
				temp = ungappedExtensionNum[r];
				ungappedExtensionNum[r] = ungappedExtensionNum[l];
				ungappedExtensionNum[l] = temp;

				temp = orderArray[r];
				orderArray[r] = orderArray[l];
				orderArray[l] = temp;
			}
		}
		--l;
		temp = ungappedExtensionNum[l];
		ungappedExtensionNum[l] = ungappedExtensionNum[nBegin];
		ungappedExtensionNum[nBegin] = temp;

		temp = orderArray[l];
		orderArray[l] = orderArray[nBegin];
		orderArray[nBegin] = temp;

		sortAlignments(ungappedExtensionNum, orderArray, nBegin, l);
		sortAlignments(ungappedExtensionNum, orderArray, r, nEnd);
	}

	return;
}

void alignments_fingGoodAlignmentsGPU(struct PSSMatrixFP **PSSMatrixFP,		//GPU buffer
									  struct PSSMatrixFP PSSMatrixFPH,
									  struct scoreMatrix *scoreMatrixp,
									  int2 **matrixBody,					//GPU buffer
									  struct sequenceDataFP **sequenceDataFP, //GPU buffer
									  unsigned char **subjectSeqs,		//GPU buffer
									  int4 totalSubSize)
{
	int alignmentsNum, ungappedExtensionsNum;
	int *startLocArray, *ungappedExtensionsNumArray, *tempUngappedNumArray;
	int *startLocArrayD, *ungappedExtensionsNumArrayD;
	int4 	*orderArray, *orderArrayD;
	int4	*bestScoreArray, *bestScoreArrayD;
	struct ungappedExtension *ungappedExtensions;
	struct ungappedExtension *ungappedExtensionsD;
	int4	*numGoodExtensions, *numGoodExtensionsD;
	int4	*numSemiGapping, *numSemiGappingD;
	struct gappedExtensionParameters parameters, *parametersD;
	int4 i;

	//struct timeval t1, t2;
	//time
	timerStart();

	//get the number of alignments and ungappedExtensions
	getAlignmentsNum(&alignmentsNum, &ungappedExtensionsNum);
	startLocArray = (int *)global_malloc(sizeof(int) * alignmentsNum);
	orderArray = (int *)global_malloc(sizeof(int) * alignmentsNum);
	ungappedExtensionsNumArray = (int *)global_malloc(sizeof(int) * alignmentsNum);
	tempUngappedNumArray = (int *)global_malloc(sizeof(int) * alignmentsNum);
	ungappedExtensions = (struct ungappedExtension *)global_malloc(sizeof(struct ungappedExtension) * ungappedExtensionsNum);
	if (startLocArray == NULL ||
		orderArray == NULL ||
		ungappedExtensionsNumArray == NULL ||
		ungappedExtensions == NULL)
	{
		printf("In findGoodAlignmentGPU, allocate buffers error!\n");
		exit (1);
	}

	bestScoreArray = (int4 *)global_malloc(sizeof(int4) * alignmentsNum);
	numGoodExtensions = (int4 *)global_malloc(sizeof(int4) * alignmentsNum);
	numSemiGapping = (int4 *)global_malloc(sizeof(int4) * alignmentsNum);
	if (bestScoreArray == NULL ||
		numGoodExtensions == NULL ||
		numSemiGapping == NULL)
	{
		printf("Allocate buffer 1 error!\n");
		exit(1);
	}

	//allocate gpu memory for alignments
	hipError_t cudaRes0, cudaRes1, cudaRes2, cudaRes3, cudaRes4;
	(void **)&bestScoreArrayD = clCreateBuffer(<context>, CL_MEM_READ_WRITE, sizeof(int4) * alignmentsNum, NULL, <errorcode>);
//	cudaRes1 = hipMalloc((void **)&bestScoreArrayD, sizeof(int4) * alignmentsNum);
	(void **)&orderArrayD = clCreateBuffer(<context>, CL_MEM_READ_WRITE, sizeof(int4) * alignmentsNum, NULL, <errorcode>);
//	cudaRes2 = hipMalloc((void **)&orderArrayD, sizeof(int4) * alignmentsNum);
	(void **)&ungappedExtensionsD = clCreateBuffer(<context>, CL_MEM_READ_WRITE, sizeof(int4) * alignmentsNum, NULL, <errorcode>);
//	cudaRes3 = hipMalloc((void **)&ungappedExtensionsD, sizeof(struct ungappedExtension) * ungappedExtensionsNum);
	if (cudaRes1 != hipSuccess ||
		cudaRes2 != hipSuccess ||
		cudaRes3 != hipSuccess)
	{ //add error checking
		printf("Allocate miscellaneous GPU memory error, %d, %d, %d\n", 
				cudaRes1,
				cudaRes2,
				cudaRes3);
		exit(1);
	}

	(void **)&startLocArrayD = clCreateBuffer(<context>, CL_MEM_READ_WRITE, sizeof(int4) * alignmentsNum, NULL, <errorcode>);
//	cudaRes0 = hipMalloc((void **)&startLocArrayD, sizeof(int4) * alignmentsNum);
	(void **)&ungappedExtensionsNumArrayD = clCreateBuffer(<context>, CL_MEM_READ_WRITE, sizeof(int4) * alignmentsNum, NULL, <errorcode>);
//	cudaRes1 = hipMalloc((void **)&ungappedExtensionsNumArrayD, sizeof(int4) * alignmentsNum);
	(void **)&parametersD = clCreateBuffer(<context>, CL_MEM_READ_WRITE, sizeof(int4) * alignmentsNum, NULL, <errorcode>);
//	cudaRes2 = hipMalloc((void **)&parametersD, sizeof(struct gappedExtensionParameters));
	(void **)&numGoodExtensionsD = clCreateBuffer(<context>, CL_MEM_READ_WRITE, sizeof(int4) * alignmentsNum, NULL, <errorcode>);
//	cudaRes3 = hipMalloc((void **)&numGoodExtensionsD, sizeof(int4) * alignmentsNum);
	(void **)&numSemiGappingD = clCreateBuffer(<context>, CL_MEM_READ_WRITE, sizeof(int4) * alignmentsNum, NULL, <errorcode>);
//	cudaRes4 = hipMalloc((void **)&numSemiGappingD, sizeof(int4) * alignmentsNum);
	if (cudaRes0 != hipSuccess ||
		cudaRes1 != hipSuccess ||
		cudaRes2 != hipSuccess ||
		cudaRes3 != hipSuccess ||
		cudaRes4 != hipSuccess)
	{
		printf("Allocate miscellaneous1 GPU memory error, %d, %d, %d, %d, %d\n", 
				cudaRes0,
				cudaRes1,
				cudaRes2,
				cudaRes3,
				cudaRes4);
		exit(1);
	}
	
	//buffers for bestRow and insertRow
	int4 *bestRowD, *insertQRowD;
	(void **)&bestRowD = clCreateBuffer(<context>, CL_MEM_READ_WRITE, sizeof(int4) * alignmentsNum, NULL, <errorcode>);
//	cudaRes0 = hipMalloc((void **)&bestRowD, sizeof(int4) * totalSubSize);
	(void **)&insertQRowD = clCreateBuffer(<context>, CL_MEM_READ_WRITE, sizeof(int4) * alignmentsNum, NULL, <errorcode>);
//	cudaRes1 = hipMalloc((void **)&insertQRowD, sizeof(int4) * totalSubSize);
	if (cudaRes0 != hipSuccess ||
		cudaRes1 != hipSuccess)
	{
		printf("Allocate row memory buffers error! %d, %d\n",
				cudaRes0,
				cudaRes1);
		exit(1);
	}

	//time
	timerEnd();
	strTime.iniTime += elapsedTime();

	//time
	timerStart();
	//copy alignments and ungaped extensions to the allocated buffer
	copyAlignmentsToBuffer(ungappedExtensions, startLocArray, ungappedExtensionsNumArray);

	//sort the alignments according to the number of ungapped extensions
	for (i = 0; i < alignmentsNum; i++)
	{
		orderArray[i] = i;
	}

	//time
	timerEnd();
	strTime.preprocessingTime += elapsedTime();

	//gettimeofday(&t1, NULL);
	//memcpy(tempUngappedNumArray, ungappedExtensionsNumArray, sizeof(int) * alignmentsNum);
	//sortAlignments(tempUngappedNumArray, orderArray, 0, alignmentsNum);
	//gettimeofday(&t2, NULL);

	//printf("sort time = %.3f\n", 1000.0 * (t2.tv_sec - t1.tv_sec) + (t2.tv_usec - t1.tv_usec)/1000.0);

	//time
	timerStart();
	//copy alignments info to GPU memory
	clEnqueueWriteBuffer(<commandQueue>, ungappedExtensionsD, CL_TRUE, 0, sizeof(struct ungappedExtension) * ungappedExtensionsNum, ungappedExtensions, 0, NULL, NULL);
//	hipMemcpy(ungappedExtensionsD, ungappedExtensions, sizeof(struct ungappedExtension) * ungappedExtensionsNum, hipMemcpyHostToDevice);
	clEnqueueWriteBuffer(<commandQueue>, numGoodExtensionsD, CL_TRUE, 0, sizeof(int4) * alignmentsNum, NULL, 0, NULL NULL);
//	hipMemset(numGoodExtensionsD, 0, sizeof(int4) * alignmentsNum);
	clEnqueueWriteBuffer(<commandQueue>, numSemiGappingD, CL_TRUE, 0, sizeof(int4) * alignmentsNum, NULL, 0, NULL, NULL);
//	hipMemset(numSemiGappingD, 0, sizeof(int4) * alignmentsNum);

	parameters.semiGappedOpenGap = parameters_semiGappedOpenGap;
	parameters.semiGappedExtendGap = parameters_semiGappedExtendGap;
	parameters.semiGappedExtensionN = parameters_semiGappedExtensionN;
	parameters.encoding_numCodes = encoding_numCodes;
	parameters.semiGappedDropoffIncrease = parameters_semiGappedDropoffIncrease;

	//copy parameters to GPU
	clEnqueueWriteBuffer(<commandQueue>, parametersD, CL_TRUE, 0, sizeof(struct gappedExtensionParameters), &parameters, 0, NULL, NULL);
//	hipMemcpy(parametersD, &parameters, sizeof(struct gappedExtensionParameters), 
			   hipMemcpyHostToDevice);

	clEnqueueWriteBuffer(<commandQueue>, startLocArrayD, CL_TRUE, 0, sizeof(int4) * alignmentsNum, startLocArray, 0, NULL, NULL);
//	hipMemcpy(startLocArrayD, startLocArray, sizeof(int4) * alignmentsNum, hipMemcpyHostToDevice);
	clEnqueueWriteBuffer(<commandQueue>, orderArrayD, CL_TRUE, 0, sizeof(int4) * alignmentsNum, orderArray, 0, NULL, NULL);
//	hipMemcpy(orderArrayD, orderArray, sizeof(int4) * alignmentsNum, hipMemcpyHostToDevice);
	clEnqueueWriteBuffer(<commandQueue>, ungappedExtensionsNumArrayD, CL_TRUE, 0, sizeof(int4) * alignmentsNum, ungappedExtensionsNumArray, 0, NULL, NULL);
//	hipMemcpy(ungappedExtensionsNumArrayD, ungappedExtensionsNumArray,
			   sizeof(int4) * alignmentsNum, hipMemcpyHostToDevice);
	//debug=================================
	//Use constant memory to store score matrix
	int scoreMatrixSize = encoding_numCodes * encoding_numCodes;
	clEnqueueWriteBuffer(<commandQueue>, scoreMatrixC, CL_TRUE, 0, sizeof(int2) * scoreMatrixSize, ((char *)scoreMatrixp->matrix) + sizeof(int2 *) * encoding_numCodes, 0, NULL, NULL);
//	hipMemcpyToSymbol(HIP_SYMBOL(scoreMatrixC), 
//					  ((char *)scoreMatrixp->matrix) + sizeof(int2 *) * encoding_numCodes, 
//					  sizeof(int2) * scoreMatrixSize);

	//Use constant memory to store query sequence
	unsigned char *tempQueryCode;
	tempQueryCode = (unsigned char *)global_malloc(sizeof(unsigned char) * (PSSMatrixFPH.length + 2));
	memcpy(&tempQueryCode[1], PSSMatrixFPH.queryCodes, sizeof(unsigned char) * PSSMatrixFPH.length);
	tempQueryCode[0] = encoding_sentinalCode;
	tempQueryCode[PSSMatrixFPH.length + 1] = encoding_sentinalCode;
	clEnqueueWriteBuffer(<commandQueue>, querySequenceC, CL_TRUE, 0, sizeof(unsigned char) * (PSSMatrixFPH.length + 2), tempQueryCode, 0, NULL, NULL;
//	hipMemcpyToSymbol(HIP_SYMBOL(querySequenceC), tempQueryCode, sizeof(unsigned char) * (PSSMatrixFPH.length + 2));
	free(tempQueryCode);

	//bind subject sequence to texture memory
	if (clEnqueueCopyImageToBuffer(<commandQueue>, *subjectSeqs, texSubjectSequences, (size_t) {0}, sizeof(unsigned char) * (totalSubSize + 2), 0, 0, NULL, NULL) != CL_SUCCESS)
//	if (hipBindTexture(NULL, texSubjectSequences, *subjectSeqs,
//		sizeof(unsigned char) * (totalSubSize + 2)) != hipSuccess)
	{
		printf("Cound not bind subject sequences to texture memory!\n");
		exit (1);
	}
	//==============================================

	//time
	timerEnd();
	strTime.copyTimeHostToDevice += elapsedTime();

	int4 blockSize = parameters_threadNum;
	int blockNum = (alignmentsNum - 1)/blockSize + 1;
//	if (blockNum < 14) //14 is the number of SMs on the Tesla C2050
//	{
//		blockSize = 64;
//		blockNum = (alignmentsNum - 1)/blockSize + 1;
//	}
	dim3 dimGrid(blockNum, 1);
	dim3 dimBlock(blockSize, 1);

	//configure L1 cache as 48KB
//This isn't exposed to OpenCL I don't think..
//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(semiGappedScoring_kernel), hipFuncCachePreferL1);
	
	//time
	timerStart();

//Need to do kernel parameter setting and enqueuing
//dimGrid and dimBlock are Workgroup and Localgroup sizes if I remember correctly
	clSetKernelArg(semiGappedScoring_kernel, 0, sizeof (<sequenceDataFP>), *sequenceDataFP);
	clSetKernelArg(semiGappedScoring_kernel, 1, sizeof(<subjectSeqs>), *subjetSeqs);
	clSetKernelArg(semiGappedScoring_kernel, 2, sizeof(PSSMatrixFP), *PSSMatrixFP);
	clSetKernelArg(semiGappedScoring_kernel, 3, sizeof(matrixBody), *matrixBody);
	clSetKernelArg(semiGappedScoring_kernel, 4, sizeof(parametersD), parametersD);
	clSetKernelArg(semiGappedScoring_kernel, 5, sizeof(startLocArrayD, startLocArrayD);
	clSetKernelArg(semiGappedScoring_kernel, 6, sizeof(ungappedExtensionsNumArrayD), ungappedExtensionsNumArrayD);
	clSetKernelArg(semiGappedScoring_kernel, 7, sizeof(alignmentsNum), alignmentsNum);
	clSetKernelArg(semiGappedScoring_kernel, 8, sizeof(ungappedExtensionsD), ungappedExtensionsD);
	clSetKernelArg(semiGappedScoring_kernel, 9, sizeof(bestScoreArrayD), bestScoreArrayD);
	clSetKernelArg(semiGappedScoring_kernel, 10, sizeof(numGoodExtensionsD), numGoodExtensionsD);
	clSetKernelArg(semiGappedScoring_kernel, 11, sizeof(numSemiGappingD), numSemiGappingD);
	clSetKernelArg(semiGappedScoring_kernel, 12, sizeof(orderArrayD), orderArrayD);
	clSetKernelArg(semiGappedScoring_kernel, 13, sizeof(bestRowD), bestRowD);
	clSetKernelArg(semiGappedScoring_kernel, 14, sizeof(insertQRowD), insertQRowD);
	clSetKernelArg(semiGappedScoring_kernel, 15, sizeof(statistics_gappedNominalDropoff), statistics_gappedNominalDropoff);
	clSetKernelArg(semiGappedScoring_kernel, 16, sizeof(blast_nominalR1cutoff), blast_nominalR1cutoff);
clEnqueueNDRangeKernel(<commandQueue>, semiGappedScoring_kernel, <workDimension>, NULL, <globalSize>, <localSize>, 0, NULL, NULL);
	
//	semiGappedScoring_kernel<<<dimGrid, dimBlock>>>(*sequenceDataFP,
//													*subjectSeqs,
//													*PSSMatrixFP,
//													*matrixBody,
//													parametersD,
//													startLocArrayD,
//													ungappedExtensionsNumArrayD,
//													alignmentsNum,
//													ungappedExtensionsD,
//													bestScoreArrayD,
//													numGoodExtensionsD,
//													numSemiGappingD,
//													orderArrayD,
//													bestRowD,
//													insertQRowD,
//													statistics_gappedNominalDropoff,
//													blast_nominalR1cutoff);
//What does this correspond to? Possibly:
//clEnqueueBarrier(<commandQueue>);
	hipDeviceSynchronize();
	//time
	timerEnd();
	strTime.kernelTime += elapsedTime();

	//time
	timerStart();

	clEnqueueReadBuffer(<commandQueue>, numGoodExtensionsD, CL_TRUE, 0, sizeof(int4) * alignmentsNum, numGoodExtensions, 0, NULL, NULL);
//	hipMemcpy(numGoodExtensions, numGoodExtensionsD, sizeof(int4) * alignmentsNum, hipMemcpyDeviceToHost);
	clEnqueueReadBuffer(<commandQueue>, numSemiGappingD, CL_TRUE, 0, sizeof(int) * alignmentsNum, numSemiGapping, 0, NULL, NULL);
//	hipMemcpy(numSemiGapping, numSemiGappingD, sizeof(int4) * alignmentsNum, hipMemcpyDeviceToHost);
	clEnqueueReadBuffer(<commandQueue>, bestScoreArrayD, CL_TRUE, 0, sizeof(int4) * alignmentsNum, bestScoreArray, 0, NULL, NULL);
//	hipMemcpy(bestScoreArray, bestScoreArrayD, sizeof(int4) * alignmentsNum, hipMemcpyDeviceToHost);
	clEnqueueReadBuffer(<commandQueue, ungappedExtensionsD, CL_TRUE, 0, sizeof(structe ungappedExtension) * ungappedExtensionsNum, ungappedExtensions, 0, NULL, NULL);
//	hipMemcpy(ungappedExtensions, ungappedExtensionsD, sizeof(struct ungappedExtension) * ungappedExtensionsNum, hipMemcpyDeviceToHost);
	
	//time
	timerEnd();
	strTime.copyTimeDeviceToHost += elapsedTime();

	//time
	timerStart();

	updataUngappedExtensions(ungappedExtensions);
	addGoodAlignment(bestScoreArray, numGoodExtensions, numSemiGapping);

	alignments_volumeAlignments[alignments_numVolumes] = alignments_alignments;
	alignments_numVolumes++;
	alignments_alignments = memBlocks_initialize(sizeof(struct alignment),
							 constants_initialAllocAlignments);

	//free memory
	free(startLocArray);
	free(ungappedExtensionsNumArray);
	free(tempUngappedNumArray);
	free(bestScoreArray);
	free(ungappedExtensions);
	free(numGoodExtensions);
	free(numSemiGapping);
	free(orderArray);

//Not so sure about this one or the other image call
	clReleaseMemObject(texSubjectSequences;
//	hipUnbindTexture(texSubjectSequences);

	clReleaseMemObject(startLocArrayD);
//	hipFree(startLocArrayD);
	clReleaseMemObject(ungappedExtensionsNumArrayD);
//	hipFree(ungappedExtensionsNumArrayD);
	clReleaseMemObject(bestScoreArrayD);
//	hipFree(bestScoreArrayD);
	clReleaseMemObject(ungappedExtensionsD);
//	hipFree(ungappedExtensionsD);
	clReleaseMemObject(numGoodExtensionsD);
//	hipFree(numGoodExtensionsD);
	clReleaseMemObject(numSemiGappingD);
//	hipFree(numSemiGappingD);
	clReleaseMemObject(orderArrayD);
//	hipFree(orderArrayD);
	clReleaseMemObject(parametersD);
//	hipFree(parametersD);
	clReleaseMemObject(bestRowD);
//	hipFree(bestRowD);
	clReleaseMemObject(insertQRowD);
//	hipFree(insertQRowD);
	//time
	timerEnd();
	strTime.postprocessingTime += elapsedTime();
}
